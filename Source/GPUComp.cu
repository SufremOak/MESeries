#include <type_traits>
#ifdef __cplusplus
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

std::is_void<class Tp>
template <typename T>
struct is_void : std::false_type {};

template <>
struct is_void<void> : std::true_type {};

template <typename T>
struct is_void<const T> : is_void<T> {};

template <typename T>
struct is_void<volatile T> : is_void<T> {};

template <typename T>
struct is_void<const volatile T> : is_void<T> {};

#endif;
